#include "hip/hip_runtime.h"


#include "CudaDLL.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/device_functions.h>
#include <math.h>
#include <float.h>
///////////////////////////////
///////////// Device specific operations
//////////////////////////


//#define IDX2R(i,j,N) (((i)*(N))+(j)) //easy way to address 2D array
__global__ void fftshift_2D(hipfftComplex *data, int arraysize, int row)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < arraysize; i += numThreads)	{
		int k=i%row;
		int j=i/row;
		
		float a = 1 - 2 * ((k + j) & 1);
		data[i].x *= a;
		data[i].y *= a;
	}
}




////////////////////////////////
///////////////////////// GPU Kernels
//////////////////////////////
__global__ void makeKernel(float* KernelPhase, int row, int column, float* ImgProperties, float MagXscaling) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float MagX = ImgProperties[1];
	float pixSize= ImgProperties[0];
	float nm = ImgProperties[2];
	float lambda = ImgProperties[3];


	float pixdxInv = MagX/pixSize*MagXscaling; // Magnification/pixSize
	float km = nm/lambda; // nm / lambda
														  
	for (int i = threadID; i < row*column; i += numThreads) {
		int dx = i%row;
		int dy = i/row; 

		float kdx = float( dx - row/2)*pixdxInv;
		float kdy = float( dy - row/2)*pixdxInv;
		float temp = km*km - kdx*kdx - kdy*kdy;
		KernelPhase[i]= (temp >= 0) ? sqrtf(temp) : 0;


		//fft shift... do once. 
		// something is off.. so will need to put the kernerl in the right location later
			

	}
}




__global__ void TransferFunction(hipfftComplex* img3Darray, float* bfpMag, float* bfpPhase, float* kPhase, float* zDist, int totalsize, int imgsize, int row)
		{
			const int numThreads = blockDim.x * gridDim.x;
			const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

			//additional counters
			for (int i = threadID; i < totalsize; i += numThreads)
			{
				int j = i / imgsize; 
				int k = i % imgsize;
				float mag = bfpMag[k];
				float phase = bfpPhase[k]+(kPhase[k]*zDist[j]); //multiply here already , absorb the 2*pi in there
				
				//add the result of above to the 3D array
				int l = k % row;
				int m = k / row;
				float fftshift = 1.0f - 2.0f * ((l + m) & 1); //only works for even
				img3Darray[i].x = mag*cosf(phase)*fftshift;
				img3Darray[i].y = mag*sinf(phase)*fftshift;
			}
		}


		__global__ void Cmplx2ReIm(hipfftComplex* cmplxArray, float* reArray, float* imgArray, int size, int imgsize, int row) {
			const int numThreads = blockDim.x * gridDim.x;
			const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
			for (int i = threadID; i < size; i += numThreads){
				int k = i/imgsize;
				int l =k%row ;
				int	m =k/row ;
				float fftshift = 1.0f - 2.0f*((l + m) & 1); //only works for even
				reArray[i] = cmplxArray[i].x *fftshift;
				imgArray[i] = cmplxArray[i].y *fftshift;

			}
		}



		////////////////////////////////////////////////
		//////////////// FUnction to compile into DLL
		////////////////////////////////////////////////



		void PropagateZslices(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutRe, float* h_ImgOutIm,
			float* zscale, int* arraySize, float* imgProperties){

			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			int row = arraySize[0];
			int column = arraySize[1];
			int zrange = arraySize[2];
			int resizeRow = arraySize[3];


			float MagXReScale = 1.0f/float(resizeRow);


			const int BlockSize = 512;
			int GridSize = 32 * 16 * 4;


			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			int numElements = row*column;
			size_t mem2darray = numElements*sizeof(float);

			float* d_kernelPhase;
			hipMalloc((void**)&d_kernelPhase, mem2darray);

			float *d_imgProperties;
			size_t sizePrp = 4 * sizeof(float);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel <<<GridSize, BlockSize,0,0 >>>(d_kernelPhase, row, column, d_imgProperties, MagXReScale);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2darray);
			hipMalloc((void**)&d_bfpPhase, mem2darray);

			hipMemcpy(d_bfpMag, h_bfpMag, mem2darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2darray, hipMemcpyHostToDevice);

			float *d_zscale;
			size_t memzsize = zrange * sizeof(float);
			hipMalloc((void**)&d_zscale, memzsize);
			hipMemcpy(d_zscale, zscale, memzsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			int size3Darray = row*column*zrange;
			size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
			hipMalloc((void**)&d_3DiFFT, mem3dsize);

			//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 

			float* d_ImgOutRe;
			float* d_ImgOutIm;
			size_t mem3dfloat = size3Darray*sizeof(float);
			hipMalloc((void**)&d_ImgOutRe, mem3dfloat);
			hipMalloc((void**)&d_ImgOutIm, mem3dfloat);

			//Execute Kernels
			TransferFunction <<<GridSize, BlockSize,0,0 >>> (d_3DiFFT, d_bfpMag , d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements, row);

			/////////////////////////////////////////////////////////////////////////////////////////
			///// Prepare batch 2D FFT plan, const declaration
			/////////////////////////////////////////////////////////////////////////////////////////
			/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
			int istride = 1; //means every element is used in the computation
			int ostride = 1; //means every element used in the computatio is output
			int idist = row*column;
			int odist = row*column;
			int inembed[] = { row,column };
			int onembed[] = { row,column };
			const int NRANK = 2;
			int n[NRANK] = { row,column };
			int BATCH = zrange;

			hipfftHandle BatchFFTPlan;

			if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
				inembed, istride, idist,// *inembed, istride, idist 
				onembed, ostride, odist,// *onembed, ostride, odist 
				HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
			{
				fprintf(stderr, "CUFFT Error: Unable to create plan\n");
				return;
			}


			//////// Execute the transform in-place
			if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
				fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
				return;
			}


			/* ////////// Execute the transform out-of-place */
			/*hipfftComplex *d_3Dimg;
			hipMalloc((void**)&d_3Dimg, mem3dsize);

			if (hipfftExecC2C(BatchFFTPlan, d_3DKernel, d_3Dimg, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
			return;
			}
			*/

			//free handle , Although might be able to reuse upon the last execution
			hipfftDestroy(BatchFFTPlan);


			///////////
			// FFT ends
			///////////

			//Kernel to transform into a LV happy readable array
			Cmplx2ReIm <<<GridSize, BlockSize,0,0 >>> (d_3DiFFT, d_ImgOutRe, d_ImgOutIm, size3Darray,numElements, row);



			//Copy device memory to host
			hipMemcpy(h_ImgOutRe, d_ImgOutRe, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_ImgOutIm, d_ImgOutIm, mem3dfloat, hipMemcpyDeviceToHost);


			//deallocate CUDA memory
			
			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_kernelPhase);
			hipFree(d_3DiFFT);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_ImgOutRe);
			hipFree(d_ImgOutIm);

		}
			

		void TestMakeKernel3D(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutRe, float* h_ImgOutIm,
			float* zscale, int* arraySize, float* imgProperties) {


			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			int row = arraySize[0];
			int column = arraySize[1];
			int zrange = arraySize[2];
			int resizeRow = arraySize[3];


			float MagXReScale = 1.0f / float(resizeRow);


			const int BlockSize = 512;
			int GridSize = 32 * 16 * 4;


			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			int numElements = row*column;
			size_t mem2darray = numElements * sizeof(float);

			float* d_kernelPhase;
			hipMalloc((void**)&d_kernelPhase, mem2darray);

			float *d_imgProperties;
			size_t sizePrp = 4 * sizeof(float);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel << <GridSize, BlockSize, 0, 0 >> >(d_kernelPhase, row, column, d_imgProperties, MagXReScale);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2darray);
			hipMalloc((void**)&d_bfpPhase, mem2darray);

			hipMemcpy(d_bfpMag, h_bfpMag, mem2darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2darray, hipMemcpyHostToDevice);

			float *d_zscale;
			size_t memzsize = zrange * sizeof(float);
			hipMalloc((void**)&d_zscale, memzsize);
			hipMemcpy(d_zscale, zscale, memzsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			int size3Darray = row*column*zrange;
			size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
			hipMalloc((void**)&d_3DiFFT, mem3dsize);

			//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 

			float* d_ImgOutRe;
			float* d_ImgOutIm;
			size_t mem3dfloat = size3Darray * sizeof(float);
			hipMalloc((void**)&d_ImgOutRe, mem3dfloat);
			hipMalloc((void**)&d_ImgOutIm, mem3dfloat);

			//Execute Kernels
			//TransferFunction << <GridSize, BlockSize, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements);

			//Kernel to transform into a LV happy readable array
			//Cmplx2ReIm << <GridSize, BlockSize, 0, 0 >> > (d_3DiFFT, d_ImgOutRe, d_ImgOutIm, size3Darray);



			//Copy device memory to host
			hipMemcpy(h_ImgOutRe, d_ImgOutRe, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_ImgOutIm, d_ImgOutIm, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_bfpPhase, d_kernelPhase, mem2darray, hipMemcpyDeviceToHost);

			//deallocate CUDA memory

			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_kernelPhase);
			hipFree(d_3DiFFT);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_ImgOutRe);
			hipFree(d_ImgOutIm);

		}
		
#include "hip/hip_runtime.h"
//add these two to get rid of intellisense errors

#include "CudaDLL.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/device_functions.h>
#include <math.h>
#include <float.h>
///////////////////////////////
///////////// Device specific operations
//////////////////////////


//#define IDX2R(i,j,N) (((i)*(N))+(j)) //easy way to address 2D array
__global__ void fftshift_2D(hipfftComplex *data, int arraysize, int row)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < arraysize; i += numThreads)	{
		int k=i%row;
		int j=i/row;
		
		float a = 1 - 2 * ((k + j) & 1);
		data[i].x *= a;
		data[i].y *= a;
	}
}

__device__ static __inline__ float cmagf2(float x, float y)
{
	float a, b, v, w, t;
	a = fabsf(x);
	b = fabsf(y);
	if (a > b) {
		v = a;
		w = b;
	}
	else {
		v = b;
		w = a;
	}
	t = w / v;
	t = 1.0f + t * t;
	t = v * sqrtf(t);
	if ((v == 0.0f) || (v > 3.402823466e38f) || (w > 3.402823466e38f)) {
		t = v + w;
	}
	return t;
}


////////////////////////////////
////////GPU Kernels
//////////////////////////////

//this kernel requires fftshift
__global__ void makeKernel(float* KernelPhase, int row, int column, float* ImgProperties, float MagXscaling) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float MagX = ImgProperties[1];
	float pixSize= ImgProperties[0];
	float nm = ImgProperties[2];
	float lambda = ImgProperties[3];


	float pixdxInv = MagX/pixSize*MagXscaling; // Magnification/pixSize
	float km = nm/lambda; // nm / lambda
														  
	for (int i = threadID; i < row*column; i += numThreads) {
		int dx = i%row;
		int dy = i/row; 

		float kdx = float( dx - row/2)*pixdxInv;
		float kdy = float( dy - row/2)*pixdxInv;
		float temp = km*km - kdx*kdx - kdy*kdy;
		KernelPhase[i]= (temp >= 0) ? (sqrtf(temp)-km) : 0;


		//This still needs quadrant swapping so this will not work in the ifft routine as is! 
		
			

	}
}


///Generates a kernel that is compatible with the non-shifted fft routine
__global__ void makeKernel_nonefftshift(float* KernelPhase, int row, int column, float* ImgProperties) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float pixSize = ImgProperties[0];
	float MagX = ImgProperties[1];
	float nmed = ImgProperties[2];
	float lambda = ImgProperties[3];
	float MagXscaling = 1/ImgProperties[4];
	float pixdxInv = MagX / pixSize*MagXscaling; // Magnification/pixSize
	float km = nmed / lambda; // nmed / lambda

	
	for (int i = threadID; i < row*column; i += numThreads) {
		int dx = i % row;
		int dy = i / row;
		
		dx= ((dx - row / 2)>0) ? (dx - row) : dx;
		dy= ((dy - row / 2)>0) ? (dy - row) : dy;
				
		float kdx = float(dx)*pixdxInv/row; //added division by row
		float kdy = float(dy)*pixdxInv/row;//added division by row
		float temp = km*km - kdx*kdx - kdy*kdy;
		KernelPhase[i] = (temp >= 0) ? (sqrtf(temp)-km) : 0;
	}
}

__global__ void makeKernelPhase(float* KernelPhase, int row, int column, float* ImgProperties) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const float pixdxInv = ImgProperties[1] / ImgProperties[0]; // Magnification/pixSize
	const float km = ImgProperties[2] / ImgProperties[3]; // nm / lambda


	for (int i = threadID; i < row*column; i += numThreads) {
		int dx = i % row;
		int dy = i / row;

		dx = ((dx - row / 2)>0) ? (dx - row) : dx;
		dy = ((dy - row / 2)>0) ? (dy - row) : dy;

		float kdx = float(dx)*pixdxInv/row;
		float kdy = float(dy)*pixdxInv/row;
		float temp = km*km - kdx*kdx - kdy*kdy;
		KernelPhase[i] = (temp >= 0) ? (sqrtf(temp)-km) : 0;
	}


}


__global__ void TransferFunction(hipfftComplex* img3Darray, float* bfpMag, float* bfpPhase, float* kPhase, float* zDist, int totalsize, int imgsize)
		{
			const int numThreads = blockDim.x * gridDim.x;
			const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

			//additional counters
			for (int i = threadID; i < totalsize; i += numThreads)
			{
				int j = i / imgsize; 
				int k = i % imgsize;
				float mag = bfpMag[k];
				float phase = bfpPhase[k]+(kPhase[k]*zDist[j]); //multiply here already , absorb the 2*pi in there
				img3Darray[i].x = mag*cosf(phase);
				img3Darray[i].y = mag*sinf(phase);
			}
		}


__global__ void Cmplx2ReIm(hipfftComplex* cmplxArray, float* reArray, float* imgArray, int size, int imgsize) {
			const int numThreads = blockDim.x * gridDim.x;
			const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
			for (int i = threadID; i < size; i += numThreads){
				int k = i/imgsize; //does this do anything????
				reArray[i] = cmplxArray[i].x;
				imgArray[i] = cmplxArray[i].y;

			}
		}

__global__ void Cmplx2Mag(hipfftComplex* cmplxArray, float* MagArray, int size, int imgsize) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads) {
		int k = i / imgsize;
		MagArray[i] = cmagf2(cmplxArray[i].x, cmplxArray[i].y);
		//imgArray[i] = cmplxArray[i].y;

	}
}



		////////////////////////////////////////////////
		//////////////// FUnction to compile into DLL
		////////////////////////////////////////////////

void GPU_Holo_v1(float* h_bfpMag, float* h_bfpPhase,
	float* h_ImgOutRe, float* h_ImgOutIm,
	float* zscale, int* arraySize, float* imgProperties) {
	
	// Declare all constants here from the array size
	// arraySize={row,column,zrange, resizeRow}
	// note that zscale has already been multiplied by 2pi, just so that C does not have to do so

	const int row = arraySize[0];
	const int column = arraySize[1];
	const int zrange = arraySize[2];
	const size_t memZsize = zrange * sizeof(float);
	const int size2Darray = row * column;
	const size_t mem2Darray = size2Darray * sizeof(float);
	const int size3Darray = row * column * zrange;
	const size_t mem3Darray = size3Darray * sizeof(float);
	const size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
	
	const int resizeRow = arraySize[3];
	const float MagXReScale = 1.0f / float(resizeRow);

	// Declare all constant regarding the Kernel execution sizes, will need to add a possibility to modify these from the LV as arguments
	const int BlockSizeAll = 512;
	const int GridSizeKernel = (size2Darray + BlockSizeAll - 1) / BlockSizeAll;
	const int GridSizeTransfer = (size3Darray/16 + BlockSizeAll - 1) / BlockSizeAll;

	/////////////////////////////////////
	/// Calculate the Propagation Kernel
	/////////////////////////////////////
		
	float* d_kernelPhase, float* d_imgProperties;
	const size_t sizePrp = 4 * sizeof(float);
	hipMalloc((void**)&d_kernelPhase, mem2Darray);
	hipMalloc((void**)&d_imgProperties, sizePrp);
	hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);
	makeKernelPhase <<< GridSizeKernel, BlockSizeAll, 0, 0 >>>(d_kernelPhase, row, column, d_imgProperties);


	//preallocate space for 3D array, this will be a bit costly but lets go ahead with it

	float* d_bfpMag,  float* d_bfpPhase, float *d_zscale;
	hipfftComplex *d_3DiFFT;
	hipMalloc((void**)&d_bfpMag, mem2Darray);
	hipMalloc((void**)&d_bfpPhase, mem2Darray);
	hipMalloc((void**)&d_zscale, memZsize);
	hipMemcpy(d_bfpMag, h_bfpMag, mem2Darray, hipMemcpyHostToDevice);
	hipMemcpy(d_bfpPhase, h_bfpPhase, mem2Darray, hipMemcpyHostToDevice);
	hipMemcpy(d_zscale, zscale, memZsize, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_3DiFFT, mem3dsize);
	
	//Execute Kernels
	TransferFunction << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, size2Darray);
	
	//deallocate CUDA memory
	hipFree(d_bfpMag);
	hipFree(d_bfpPhase);
	hipFree(d_zscale);
	hipFree(d_imgProperties);
	hipFree(d_kernelPhase);

	//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 
	// temporarily removed ... as the copy could be done in a single pass!
	float* d_ImgOutRe, float* d_ImgOutIm;
	hipMalloc((void**)&d_ImgOutRe, mem3Darray);
	hipMalloc((void**)&d_ImgOutIm, mem3Darray);

	/////////////////////////////////////////////////////////////////////////////////////////
	///// Prepare batch 2D FFT plan, const declaration , should be just called a function
	/////////////////////////////////////////////////////////////////////////////////////////
	/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
	int istride = 1; //means every element is used in the computation
	int ostride = 1; //means every element used in the computatio is output
	int idist = row*column;
	int odist = row*column;
	int inembed[] = { row,column };
	int onembed[] = { row,column };
	const int NRANK = 2;
	int n[NRANK] = { row,column };
	int BATCH = zrange;

	hipfftHandle BatchFFTPlan;
	if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
		inembed, istride, idist,// *inembed, istride, idist 
		onembed, ostride, odist,// *onembed, ostride, odist 
		HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return;
	}
	
	//////// Execute the transform in-place
	if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
		return;
	}

	//free handle , Although might be able to reuse upon the last execution
	hipfftDestroy(BatchFFTPlan);


	///////////
	// FFT ends
	///////////

	//Kernel to transform into a LV happy readable array
	Cmplx2ReIm <<<GridSizeTransfer, BlockSizeAll, 0, 0 >>> (d_3DiFFT, d_ImgOutRe, d_ImgOutIm, size3Darray, size2Darray);
	hipFree(d_3DiFFT);
	
	hipMemcpy(h_ImgOutRe, d_ImgOutRe, mem3Darray, hipMemcpyDeviceToHost);
	hipMemcpy(h_ImgOutIm, d_ImgOutIm, mem3Darray, hipMemcpyDeviceToHost);
	hipFree(d_ImgOutRe);
	hipFree(d_ImgOutIm);

}


void GPU_Holo_v2(float* h_bfpMag, float* h_bfpPhase,
	float* h_ImgOutAmp, float* zscale, int* arraySize, float* imgProperties) {

	// Declare all constants here from the array size
	// arraySize={row,column,zrange, resizeRow}
	// note that zscale has already been multiplied by 2pi, just so that C does not have to do so

	const int row = arraySize[0];
	const int column = arraySize[1];
	const int zrange = arraySize[2];
	const size_t memZsize = zrange * sizeof(float);
	const int size2Darray = row * column;
	const size_t mem2Darray = size2Darray * sizeof(float);
	const int size3Darray = row * column * zrange;
	const size_t mem3Darray = size3Darray * sizeof(float);
	const size_t mem3dsize = size3Darray * sizeof(hipfftComplex);

	const int resizeRow = arraySize[3];
	const float MagXReScale = 1.0f / float(resizeRow);

	// Declare all constant regarding the Kernel execution sizes, will need to add a possibility to modify these from the LV as arguments
	const int BlockSizeAll = 512;
	const int GridSizeKernel = (size2Darray + BlockSizeAll - 1) / BlockSizeAll;
	const int GridSizeTransfer = (size3Darray / 16 + BlockSizeAll - 1) / BlockSizeAll;

	/////////////////////////////////////
	/// Calculate the Propagation Kernel
	/////////////////////////////////////

	float* d_kernelPhase, float* d_imgProperties;
	const size_t sizePrp = 4 * sizeof(float);
	hipMalloc((void**)&d_kernelPhase, mem2Darray);
	hipMalloc((void**)&d_imgProperties, sizePrp);
	hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);
	makeKernelPhase << < GridSizeKernel, BlockSizeAll, 0, 0 >> >(d_kernelPhase, row, column, d_imgProperties);


	//preallocate space for 3D array, this will be a bit costly but lets go ahead with it

	float* d_bfpMag, float* d_bfpPhase, float *d_zscale;
	hipfftComplex *d_3DiFFT;
	hipMalloc((void**)&d_bfpMag, mem2Darray);
	hipMalloc((void**)&d_bfpPhase, mem2Darray);
	hipMalloc((void**)&d_zscale, memZsize);
	hipMemcpy(d_bfpMag, h_bfpMag, mem2Darray, hipMemcpyHostToDevice);
	hipMemcpy(d_bfpPhase, h_bfpPhase, mem2Darray, hipMemcpyHostToDevice);
	hipMemcpy(d_zscale, zscale, memZsize, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_3DiFFT, mem3dsize);

	//Execute Kernels
	TransferFunction << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, size2Darray);

	//deallocate CUDA memory
	hipFree(d_bfpMag);
	hipFree(d_bfpPhase);
	hipFree(d_zscale);
	hipFree(d_imgProperties);
	hipFree(d_kernelPhase);

	//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 
	// temporarily removed ... as the copy could be done in a single pass!
	float* d_ImgOutAmp;
	hipMalloc((void**)&d_ImgOutAmp, mem3Darray);

	/////////////////////////////////////////////////////////////////////////////////////////
	///// Prepare batch 2D FFT plan, const declaration , should be just called a function
	/////////////////////////////////////////////////////////////////////////////////////////
	/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
	int istride = 1; //means every element is used in the computation
	int ostride = 1; //means every element used in the computatio is output
	int idist = row*column;
	int odist = row*column;
	int inembed[] = { row,column };
	int onembed[] = { row,column };
	const int NRANK = 2;
	int n[NRANK] = { row,column };
	int BATCH = zrange;

	hipfftHandle BatchFFTPlan;
	if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
		inembed, istride, idist,// *inembed, istride, idist 
		onembed, ostride, odist,// *onembed, ostride, odist 
		HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return;
	}

	//////// Execute the transform in-place
	if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
		return;
	}

	//free handle , Although might be able to reuse upon the last execution
	hipfftDestroy(BatchFFTPlan);


	///////////
	// FFT ends
	///////////

	//Kernel to transform into a LV happy readable array
	Cmplx2Mag << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_ImgOutAmp, size3Darray, size2Darray);
	hipFree(d_3DiFFT);

	hipMemcpy(h_ImgOutAmp, d_ImgOutAmp, mem3Darray, hipMemcpyDeviceToHost);
	hipFree(d_ImgOutAmp);

}

void PropagateZslices(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutRe, float* h_ImgOutIm,
			float* zscale, int* arraySize, float* imgProperties){

			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			int row = arraySize[0];
			int column = arraySize[1];
			int zrange = arraySize[2];
			int resizeRow = arraySize[3];
			float MagXReScale = 1.0f/float(resizeRow);
			
			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			int numElements = row*column;
			size_t mem2darray = numElements*sizeof(float);

			const int BlockSizeAll = 512;
			int GridSizeKernel = (numElements + BlockSizeAll-1)/BlockSizeAll;


			float* d_kernelPhase;
			hipMalloc((void**)&d_kernelPhase, mem2darray);

			float *d_imgProperties;
			size_t sizePrp = 4 * sizeof(float);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel_nonefftshift <<<GridSizeKernel, BlockSizeAll,0,0 >>>(d_kernelPhase, row, column, d_imgProperties);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2darray);
			hipMalloc((void**)&d_bfpPhase, mem2darray);

			hipMemcpy(d_bfpMag, h_bfpMag, mem2darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2darray, hipMemcpyHostToDevice);

			float *d_zscale;
			size_t memzsize = zrange * sizeof(float);
			hipMalloc((void**)&d_zscale, memzsize);
			hipMemcpy(d_zscale, zscale, memzsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			int size3Darray = row*column*zrange;
			size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
			hipMalloc((void**)&d_3DiFFT, mem3dsize);

			//Execute Kernels
			int GridSizeTransfer = (numElements*zrange/16+BlockSizeAll-1)/BlockSizeAll;
			TransferFunction <<<GridSizeTransfer, BlockSizeAll,0,0 >>> (d_3DiFFT, d_bfpMag , d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements);
			
			//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 
			// temporarily removed ... as the copy could be done in a single pass!
			float* d_ImgOutRe;
			float* d_ImgOutIm;
			size_t mem3dfloat = size3Darray*sizeof(float);
			hipMalloc((void**)&d_ImgOutRe, mem3dfloat);
			hipMalloc((void**)&d_ImgOutIm, mem3dfloat);


			/////////////////////////////////////////////////////////////////////////////////////////
			///// Prepare batch 2D FFT plan, const declaration
			/////////////////////////////////////////////////////////////////////////////////////////
			/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
			int istride = 1; //means every element is used in the computation
			int ostride = 1; //means every element used in the computatio is output
			int idist = row*column;
			int odist = row*column;
			int inembed[] = { row,column };
			int onembed[] = { row,column };
			const int NRANK = 2;
			int n[NRANK] = { row,column };
			int BATCH = zrange;

			hipfftHandle BatchFFTPlan;

			if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
				inembed, istride, idist,// *inembed, istride, idist 
				onembed, ostride, odist,// *onembed, ostride, odist 
				HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
			{
				fprintf(stderr, "CUFFT Error: Unable to create plan\n");
				return;
			}


			//////// Execute the transform in-place
			if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
				fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
				return;
			}
			
			//free handle , Although might be able to reuse upon the last execution
			hipfftDestroy(BatchFFTPlan);


			///////////
			// FFT ends
			///////////

			//Kernel to transform into a LV happy readable array
			Cmplx2ReIm <<<GridSizeTransfer, BlockSizeAll,0,0 >>> (d_3DiFFT, d_ImgOutRe, d_ImgOutIm, size3Darray,numElements);
			
			//Copy device memory to hosts
					
			hipMemcpy(h_ImgOutRe,d_ImgOutRe, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_ImgOutIm,d_ImgOutIm, mem3dfloat, hipMemcpyDeviceToHost);


			//deallocate CUDA memory
			
			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_kernelPhase);
			hipFree(d_3DiFFT);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_ImgOutRe);
			hipFree(d_ImgOutIm);

		}



void PropagateZ_ReturnMagnitude(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutMag, float* zscale, int* arraySize, float* imgProperties) {

			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			int row = arraySize[0];
			int column = arraySize[1];
			int zrange = arraySize[2];
			int resizeRow = arraySize[3];
			float MagXReScale = 1.0f / float(resizeRow);
			
			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			int numElements = row*column;
			size_t mem2darray = numElements * sizeof(float);

			const int BlockSizeAll = 512;
			int GridSizeKernel = (numElements + BlockSizeAll - 1) / BlockSizeAll;


			float* d_kernelPhase;
			hipMalloc((void**)&d_kernelPhase, mem2darray);

			float *d_imgProperties;
			size_t sizePrp = 4 * sizeof(float);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel_nonefftshift << <GridSizeKernel, BlockSizeAll, 0, 0 >> >(d_kernelPhase, row, column, d_imgProperties);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2darray);
			hipMalloc((void**)&d_bfpPhase, mem2darray);

			hipMemcpy(d_bfpMag, h_bfpMag, mem2darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2darray, hipMemcpyHostToDevice);

			float *d_zscale;
			size_t memzsize = zrange * sizeof(float);
			hipMalloc((void**)&d_zscale, memzsize);
			hipMemcpy(d_zscale, zscale, memzsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			int size3Darray = row*column*zrange;
			size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
			hipMalloc((void**)&d_3DiFFT, mem3dsize);

			//Execute Kernels
			int GridSizeTransfer = (numElements*zrange / 16 + BlockSizeAll - 1) / BlockSizeAll;
			TransferFunction << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements);

			//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 
			// temporarily removed ... as the copy could be done in a single pass!
			float* d_ImgOutMag;
			//float* d_ImgOutIm;
			size_t mem3dfloat = size3Darray * sizeof(float);
			hipMalloc((void**)&d_ImgOutMag, mem3dfloat);
			//hipMalloc((void**)&d_ImgOutIm, mem3dfloat);


			/////////////////////////////////////////////////////////////////////////////////////////
			///// Prepare batch 2D FFT plan, const declaration
			/////////////////////////////////////////////////////////////////////////////////////////
			/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
			int istride = 1; //means every element is used in the computation
			int ostride = 1; //means every element used in the computatio is output
			int idist = row*column;
			int odist = row*column;
			int inembed[] = { row,column };
			int onembed[] = { row,column };
			const int NRANK = 2;
			int n[NRANK] = { row,column };
			int BATCH = zrange;

			hipfftHandle BatchFFTPlan;

			if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
				inembed, istride, idist,// *inembed, istride, idist 
				onembed, ostride, odist,// *onembed, ostride, odist 
				HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
			{
				fprintf(stderr, "CUFFT Error: Unable to create plan\n");
				return;
			}


			//////// Execute the transform in-place
			if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
				fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
				return;
			}

			//free handle , Although might be able to reuse upon the last execution
			hipfftDestroy(BatchFFTPlan);


			///////////
			// FFT ends
			///////////

			//Kernel to transform into a LV happy readable array
			Cmplx2Mag << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_ImgOutMag, size3Darray, numElements);

			//Copy device memory to hosts

			hipMemcpy(h_ImgOutMag, d_ImgOutMag, mem3dfloat, hipMemcpyDeviceToHost);
			//hipMemcpy(h_ImgOutIm, d_ImgOutIm, mem3dfloat, hipMemcpyDeviceToHost);


			//deallocate CUDA memory

			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_kernelPhase);
			hipFree(d_3DiFFT);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_ImgOutMag);
			//hipFree(d_ImgOutIm);

		}
			

void ReturnMagnitudeZStack2(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutMag, float* zscale, int* arraySize, float* imgProperties, int* GPUspecs) {

			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			const int row = arraySize[0];
			const int column = arraySize[1];
			const int zrange = arraySize[2];
			
			const int numElements = row*column;
			const int size3Darray = row * column*zrange;

			const size_t memZsize = zrange * sizeof(float);
			const size_t mem2Darray = numElements * sizeof(float);
			const size_t mem3Dsize = size3Darray * sizeof(hipfftComplex);
			const size_t mem3Darray = size3Darray * sizeof(float);
			const size_t sizePrp = 5 * sizeof(float);
						

			//Declare all constants regarding Kernel execution sizes
			const int BlockSizeAll = 512; //GPUspecs[0];
			const int GridSizeKernel = (numElements + BlockSizeAll - 1) / BlockSizeAll;
			const int GridSizeTransfer = (size3Darray / 16 + BlockSizeAll - 1) / BlockSizeAll;

			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			float* d_kernelPhase;
			float *d_imgProperties;
			hipMalloc((void**)&d_kernelPhase, mem2Darray);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel_nonefftshift <<<GridSizeKernel, BlockSizeAll, 0, 0 >> >(d_kernelPhase, row, column, d_imgProperties);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2Darray);
			hipMalloc((void**)&d_bfpPhase, mem2Darray);
			hipMemcpy(d_bfpMag, h_bfpMag, mem2Darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2Darray, hipMemcpyHostToDevice);

			float *d_zscale;
			hipMalloc((void**)&d_zscale, memZsize);
			hipMemcpy(d_zscale, zscale, memZsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			hipMalloc((void**)&d_3DiFFT, mem3Dsize);

			//Execute Kernels
			TransferFunction << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements);
			
			//deallocate CUDA memory
			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_kernelPhase);


			//Allocate cuda memory for 3D FFT
			float* d_ImgOutMag;
			hipMalloc((void**)&d_ImgOutMag, mem3Darray);


			/////////////////////////////////////////////////////////////////////////////////////////
			///// Prepare batch 2D FFT plan, const declaration
			/////////////////////////////////////////////////////////////////////////////////////////
			/* Create a batched 2D plan, or batch FFT , need to declare when each image begins! */
			int istride = 1; //means every element is used in the computation
			int ostride = 1; //means every element used in the computatio is output
			int idist = row*column;
			int odist = row*column;
			int inembed[] = { row,column };
			int onembed[] = { row,column };
			const int NRANK = 2;
			int n[NRANK] = { row,column };
			int BATCH = zrange;

			hipfftHandle BatchFFTPlan;

			if (hipfftPlanMany(&BatchFFTPlan, NRANK, n,
				inembed, istride, idist,// *inembed, istride, idist 
				onembed, ostride, odist,// *onembed, ostride, odist 
				HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
			{
				fprintf(stderr, "CUFFT Error: Unable to create plan\n");
				return;
			}


			//////// Execute the transform in-place
			if (hipfftExecC2C(BatchFFTPlan, d_3DiFFT, d_3DiFFT, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
				fprintf(stderr, "CUFFT Error: Failed to execute plan\n");
				return;
			}

			//free handle , Although might be able to reuse upon the last execution
			hipfftDestroy(BatchFFTPlan);


			///////////
			// FFT ends
			///////////

			//Kernel to transform into a LV happy readable array
			Cmplx2Mag << <GridSizeTransfer, BlockSizeAll, 0, 0 >> > (d_3DiFFT, d_ImgOutMag, size3Darray, numElements);

			//Copy device memory to hosts
			hipMemcpy(h_ImgOutMag, d_ImgOutMag, mem3Darray, hipMemcpyDeviceToHost);


			//deallocate CUDA memory

		
			
			hipFree(d_3DiFFT);
			hipFree(d_ImgOutMag);

		}




void TestMakeKernel3D(float* h_bfpMag, float* h_bfpPhase,
			float* h_ImgOutRe, float* h_ImgOutIm,
			float* zscale, int* arraySize, float* imgProperties) {


			//Extract the size of the 2D and 3D arrays, and their respect allocation sizes
			int row = arraySize[0];
			int column = arraySize[1];
			int zrange = arraySize[2];
			int resizeRow = arraySize[3];


			float MagXReScale = 1.0f / float(resizeRow);


			const int BlockSize = 512;
			int GridSize = 32 * 16 * 4;


			//////////////////////////////////////////////////
			//transfer data from host memory to GPU 
			//// idea is to avoid an expensive c++ allocation and copying values into a complex array format
			////// Almost thinking of calculating the whole Kernel in the device to avoid 2 device transfers!

			int numElements = row*column;
			size_t mem2darray = numElements * sizeof(float);

			float* d_kernelPhase;
			hipMalloc((void**)&d_kernelPhase, mem2darray);

			float *d_imgProperties;
			size_t sizePrp = 4 * sizeof(float);
			hipMalloc((void**)&d_imgProperties, sizePrp);
			hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);

			makeKernel << <GridSize, BlockSize, 0, 0 >> >(d_kernelPhase, row, column, d_imgProperties, MagXReScale);

			float* d_bfpMag;
			float* d_bfpPhase;
			hipMalloc((void**)&d_bfpMag, mem2darray);
			hipMalloc((void**)&d_bfpPhase, mem2darray);

			hipMemcpy(d_bfpMag, h_bfpMag, mem2darray, hipMemcpyHostToDevice);
			hipMemcpy(d_bfpPhase, h_bfpPhase, mem2darray, hipMemcpyHostToDevice);

			float *d_zscale;
			size_t memzsize = zrange * sizeof(float);
			hipMalloc((void**)&d_zscale, memzsize);
			hipMemcpy(d_zscale, zscale, memzsize, hipMemcpyHostToDevice);

			//preallocate space for 3D array, this will be a bit costly but lets go ahead with it
			hipfftComplex *d_3DiFFT;
			int size3Darray = row*column*zrange;
			size_t mem3dsize = size3Darray * sizeof(hipfftComplex);
			hipMalloc((void**)&d_3DiFFT, mem3dsize);

			//given that LV does not accept the cmplx number array format as any I/O I need to transform the cmplx 3D array into re and im. 

			float* d_ImgOutRe;
			float* d_ImgOutIm;
			size_t mem3dfloat = size3Darray * sizeof(float);
			hipMalloc((void**)&d_ImgOutRe, mem3dfloat);
			hipMalloc((void**)&d_ImgOutIm, mem3dfloat);

			//Execute Kernels
			//TransferFunction << <GridSize, BlockSize, 0, 0 >> > (d_3DiFFT, d_bfpMag, d_bfpPhase, d_kernelPhase, d_zscale, size3Darray, numElements);

			//Kernel to transform into a LV happy readable array
			//Cmplx2ReIm << <GridSize, BlockSize, 0, 0 >> > (d_3DiFFT, d_ImgOutRe, d_ImgOutIm, size3Darray);



			//Copy device memory to host
			hipMemcpy(h_ImgOutRe, d_ImgOutRe, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_ImgOutIm, d_ImgOutIm, mem3dfloat, hipMemcpyDeviceToHost);
			hipMemcpy(h_bfpPhase, d_kernelPhase, mem2darray, hipMemcpyDeviceToHost);

			//deallocate CUDA memory

			hipFree(d_bfpMag);
			hipFree(d_bfpPhase);
			hipFree(d_kernelPhase);
			hipFree(d_3DiFFT);
			hipFree(d_zscale);
			hipFree(d_imgProperties);
			hipFree(d_ImgOutRe);
			hipFree(d_ImgOutIm);

		}
		
#include "hip/hip_runtime.h"
#include "CudaDLL.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/device_functions.h>
#include <math.h>
#include <float.h>
///////////////////////////////
///////////// Device specific operations
//////////////////////////


__global__ void real2complex(float *dataIn, hipfftComplex *dataOut, int arraysize)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < arraysize; i += numThreads) {
		dataOut[i].x = dataIn[i];
		dataOut[i].y = 0.0f;
	}
	
}


__global__ void C2R(hipfftComplex* cmplxArray, float* reArray, float* imgArray, int size) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads) {
		reArray[i] = cmplxArray[i].x;
		imgArray[i] = cmplxArray[i].y;

	}
}

__global__ void TiltCorrection(hipfftComplex* imgData, float* imgProp, int row, int column) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = row*column;
	const float kdx = imgProp[0];
	const float kdy = imgProp[1];
	const float kdr = imgProp[2];

	for (int i = threadID; i < size; i += numThreads) {
	
	}
}

__global__ void FrequencyFilter(hipfftComplex* BFP, hipfftComplex* GradBFP, float* imgProp, int row, int column, BOOLEAN Top) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = row*column;
	const float kdx = Top ? imgProp[0] : imgProp[1];
	const float kdy = Top ? imgProp[1] : imgProp[0];
	const float kdr = imgProp[2];

	for (int i = threadID; i < size; i += numThreads) {
		int idx = i % row;
		int idy = i / row;

		/* represents the mask for bandpass frequency filtering*/
		int dx = (idx < (row / 2)) ? idx : (idx - row);
		int dy = (idy < (row / 2)) ? idy : (idy - row);
		float temp = kdr*kdr - dx*dx - dy*dy;

		/*Find the index to shift the BFP by!, */
		if (idx < (row/2)) {
			//idx = (kdx>0)? idx+kdx : row + (idx - kdx);
			idx = idx + kdx;
		}
		else {
			//idx = (-dx > kdx) ? idx+kdx  : row + (-dx - kdx);
			idx = dx + kdx;
		}

		if (idy < (row/2)){
			//idy = (idy > kdy) ? idy + kdy : row + (idy - kdy);
			idy = idy + kdy;
		}
		else {
			//idy = (-dy > kdy) ? dy + kdy : row + (-dy - kdy);
			idy = dy + kdy;
		}

		//;
		//;

		GradBFP[i].x = (temp>=0) ? BFP[idx + idy*row].x : 0;
		GradBFP[i].y = (temp>=0) ? BFP[idx + idy*row].y : 0;
		
	}
}
///////////////////////
//////////////// Executable functions 
///////////////////////


void ExtractGradients(float* h_rawImg, int* arraySize, float* imgProperties,
	float* h_ImgDxOutRe, float* h_ImgDxOutIm,
	float* h_ImgDyOutRe, float* h_ImgDyOutIm) {
	
//Declare constants
	const int row = arraySize[0];
	const int column = arraySize[1];
	const int zrange = 1; // in this case Matz is only doing one image at a time
	const int imgpropsize = arraySize[2];
	const size_t size2Darray = row*column;
	const size_t mem2Darray = size2Darray * sizeof(float);
	const size_t mem2DFFTsize = size2Darray * sizeof(hipfftComplex);

 // Declare all constant regarding the Kernel execution sizes, will need to add a possibility to modify these from the LV as arguments
	const int BlockSizeAll = 512;
	const int GridSizeKernel = (size2Darray + BlockSizeAll - 1) / BlockSizeAll;

// Copy Raw Img and spatial filtering constants to GPU device
	float* d_rawImg, float* d_imgProperties;
	const size_t sizePrp = imgpropsize * sizeof(float);
	hipMalloc((void**)&d_rawImg, mem2Darray);
	hipMemcpy(d_rawImg, h_rawImg, mem2Darray, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_imgProperties, sizePrp);
	hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);
	

//Img memory allocations on the GPU to hold BFP and derivates of X and Y
	hipfftComplex *d_BFP;
	hipfftComplex *d_GradDx;
	hipfftComplex *d_GradDy;
	hipMalloc((void**)&d_BFP, mem2DFFTsize);
	hipMalloc((void**)&d_GradDx, mem2DFFTsize);
	hipMalloc((void**)&d_GradDy, mem2DFFTsize);
	
	

 /////////////////////////////////////////////////////////////////////////////////////////
 ///// Prepare batch 2D FFT plan, const declaration
 /////////////////////////////////////////////////////////////////////////////////////////
	
	int istride = 1; //means every element is used in the computation
	int ostride = 1; //means every element used in the computatio is output
	int idist = row*column;
	int odist = row*column;
	int inembed[] = { row,column };
	int onembed[] = { row,column };
	const int NRANK = 2;
	int n[NRANK] = { row,column };
	int BATCH = zrange;

	hipfftHandle SingleFFTPlan;
		if (hipfftPlanMany(&SingleFFTPlan, NRANK, n,
		inembed, istride, idist,// *inembed, istride, idist 
		onembed, ostride, odist,// *onembed, ostride, odist 
		HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return;
	}

	

	// Convert d-raw img into a complex number!
		real2complex <<<GridSizeKernel, BlockSizeAll, 0, 0 >>>(d_rawImg, d_BFP, size2Darray);

	/// Execute FFT transform in-place to go into kspace, 
		hipfftExecC2C(SingleFFTPlan, d_BFP, d_BFP, HIPFFT_FORWARD);
		//so far so good up to here

	/// Extract gradients in X and Y, frequency filtering 
		BOOLEAN Top = 1;
		FrequencyFilter <<<GridSizeKernel, BlockSizeAll, 0, 0 >>> (d_BFP, d_GradDy, d_imgProperties, row, column, Top);
		Top = 0;
		FrequencyFilter <<<GridSizeKernel, BlockSizeAll, 0, 0 >>> (d_BFP, d_GradDx, d_imgProperties, row, column, Top);

		//Seems that i have a problem here!


	/// Inverse FFT in-place for each of the gradients
		//hipfftExecC2C(SingleFFTPlan, d_GradDx, d_GradDx, HIPFFT_BACKWARD);
		//hipfftExecC2C(SingleFFTPlan, d_GradDy, d_GradDy, HIPFFT_BACKWARD);

	//free handle , Although might be able to reuse upon the last execution
		hipfftDestroy(SingleFFTPlan);
	
// Copy FFT result to output

		float *d_ImgDxOutRe; 
		float *d_ImgDxOutIm;
		float *d_ImgDyOutRe;
		float *d_ImgDyOutIm;
		hipMalloc((void**)&d_ImgDxOutRe, mem2Darray);
		hipMalloc((void**)&d_ImgDxOutIm, mem2Darray);
		hipMalloc((void**)&d_ImgDyOutRe, mem2Darray);
		hipMalloc((void**)&d_ImgDyOutIm, mem2Darray);
		

		//C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_GradDx, d_ImgDxOutRe, d_ImgDxOutIm, size2Darray);
		C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_BFP, d_ImgDxOutRe, d_ImgDxOutIm, size2Darray);
		hipFree(d_GradDx);
		C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_GradDy, d_ImgDyOutRe, d_ImgDyOutIm, size2Darray);
		hipFree(d_GradDy);

		hipMemcpy(h_ImgDxOutRe, d_ImgDxOutRe, mem2Darray, hipMemcpyDeviceToHost);
		hipMemcpy(h_ImgDxOutIm, d_ImgDxOutIm, mem2Darray, hipMemcpyDeviceToHost);
		hipFree(d_ImgDxOutRe);
		hipFree(d_ImgDxOutIm);

		hipMemcpy(h_ImgDyOutRe, d_ImgDyOutRe, mem2Darray, hipMemcpyDeviceToHost);
		hipMemcpy(h_ImgDyOutIm, d_ImgDyOutIm, mem2Darray, hipMemcpyDeviceToHost);
		hipFree(d_ImgDyOutRe);
		hipFree(d_ImgDyOutIm);

		//exporting is correct
		//d_ImgdxOutRe
	///////////
	// FFT ends
	///////////

}


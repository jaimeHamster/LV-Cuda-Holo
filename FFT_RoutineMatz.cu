#include "hip/hip_runtime.h"
#include "CudaDLL.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/device_functions.h>
#include <math.h>
#include <float.h>
///////////////////////////////
///////////// Device specific operations
//////////////////////////

//#define sign(a) ((a) > 0 ? +1 : ((a) < 0 ? -1 : 0));

///
///
#define BLOCKSIZE_CART2POL	256

template <class T>
__global__ void Cartesian2PolarKernel(const T * __restrict__ d_x, const T * __restrict__ d_y, T * __restrict__ d_rho, T * __restrict__ d_theta,
	const int N, const T a) {

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N) {
		d_rho[tid] = a * hypot(d_x[tid], d_y[tid]);
		d_theta[tid] = atan2(d_y[tid], d_x[tid]);
	}

}

/*******************************************************/
/* CARTESIAN TO POLAR COORDINATES TRANSFORMATION - GPU */
/*******************************************************/
//template <class T>
//thrust::pair<T *,T *> Cartesian2Polar(const T * __restrict__ d_x, const T * __restrict__ d_y, const int N, const T a) {
//
//	T *d_rho;	gpuErrchk(hipMalloc((void**)&d_rho,   N * sizeof(T)));
//	T *d_theta; gpuErrchk(hipMalloc((void**)&d_theta, N * sizeof(T)));
//
//	Cartesian2PolarKernel<<<iDivUp(N, BLOCKSIZE_CART2POL), BLOCKSIZE_CART2POL>>>(d_x, d_y, d_rho, d_theta, N, a);
//#ifdef DEBUG
//	gpuErrchk(hipPeekAtLastError());
//	gpuErrchk(hipDeviceSynchronize());
//#endif
//
//	return thrust::make_pair(d_rho, d_theta);
//}
//
//template thrust::pair<float  *, float  *>  Cartesian2Polar<float>  (const float  *, const float  *, const int, const float);
//template thrust::pair<double *, double *>  Cartesian2Polar<double> (const double *, const double *, const int, const double);



__global__ void real2complex(float *dataIn, hipfftComplex *dataOut, int arraysize)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < arraysize; i += numThreads) {
		dataOut[i].x = dataIn[i];
		dataOut[i].y = 0.0f;
	}
	
}


__global__ void C2R(hipfftComplex* cmplxArray, float* reArray, float* imgArray, int size) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads) {
		reArray[i] = cmplxArray[i].x;
		imgArray[i] = cmplxArray[i].y;

	}
}


__global__ void ExtractGradsBFP(hipfftComplex* BFP,
	hipfftComplex* GradxBFP, hipfftComplex* GradyBFP, hipfftComplex* DC_BFP,
	int* imgProp, int row, int column) 
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = row*column;
	const int kdx = imgProp[0];
	const int kdy = imgProp[1];
	const int kdr = imgProp[2];

	//only seems to work for images with an odd number of rows and columns!
	//breaks down for negative kdx and kdy

	for (int i = threadID; i < size; i += numThreads) {
		int idx = i % row;
		int idy = i / row;
		
		//bunch of dummy variables to avoid overwriting
		int idx2= idx;
		int idy2= idy;
		int idx1 = idx;
		int idy1 = idy;
		
		/* represents the mask for bandpass frequency filtering*/
		int dx = (idx < (row / 2)) ? idx : (idx - row);
		int dy = (idy < (row / 2)) ? idy : (idy - row);
		float temp = kdr*kdr - dx*dx - dy*dy;

		//declare a bunch of dummy variables to hold the different indices
		int tempxx = idx + kdx;
		int tempyy = idy + kdy;
		int tempxy = idx + kdy;
		int tempyx = idy + kdx;

		// no need for the case of tempx>row/2 since this continues as is!

		if (idx < (row / 2)){
			idx1 = (tempxx < 0) ? row + tempxx : tempxx;
			idx2 = (tempxy < 0) ? row + tempxy : tempxy;
		}
		else{
			idx1 = (tempxx < row) ? tempxx  :dx + kdx ;
			idx2 = (tempxy < row) ? tempxy : dx + kdy ;
		}

		if (idy < (row / 2)) {
			idy1 = (tempyy < 0) ? row + tempyy : tempyy;
			idy2 = (tempyx < 0) ? row + tempyx : tempyx;
		}
		else {
			idy1 = (tempyy < row) ?  tempyy : dy + kdy;
			idy2 = (tempyx < row) ? tempyx : dy + kdx;
		}

		
		GradyBFP[i].x = (temp >= 0) ? BFP[idx1 + idy1*row].x : 0;
		GradyBFP[i].y = (temp >= 0) ? BFP[idx1 + idy1*row].y : 0;
		GradxBFP[i].x = (temp >= 0) ? BFP[idx2 + idy2*row].x : 0;
		GradxBFP[i].y = (temp >= 0) ? BFP[idx2 + idy2*row].y : 0;
		DC_BFP[i].x = (temp >= 0) ? BFP[i].x : 0;
		DC_BFP[i].y = (temp >= 0) ? BFP[i].y : 0;

	}
}


__global__ void ExtractGradsBFP_Optimised(hipfftComplex* BFP,
	hipfftComplex* GradAll, int* imgProp, int row, int column)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = row*column*3; // three sets of arrays!
	const int kdr = imgProp[2];
	int kdx;
	int kdy;

	//only seems to work for images with an odd number of rows and columns!
	//breaks down for negative kdx and kdy

	for (int i = threadID; i < size; i += numThreads) {
		int k = i % (row*column);
		int inum = i/ (row*column);
		int idx = k % row;
		int idy = k / row;
		
		if (inum == 0) {
			kdx = imgProp[0];
			kdy = imgProp[1];
		}
		else if (inum == 1) {
			kdx = imgProp[1];
			kdy = imgProp[0];
		}

		else {
			kdx = 0;
			kdy = 0;
		}
				
		/* represents the mask for bandpass frequency filtering*/
		int dx = (idx < (row / 2)) ? idx : (idx - row);
		int dy = (idy < (row / 2)) ? idy : (idy - row);
		float temp = kdr*kdr - dx*dx - dy*dy;

		//declare a bunch of dummy variables to hold the different indices
		int tempxx = idx + kdx;
		int tempyy = idy + kdy;
		idx = (tempxx < 0) ? row + tempxx : (tempxx < row) ? tempxx : dx + kdx;
		idy = (tempyy < 0) ? row + tempyy : (tempyy < row) ? tempyy : dy + kdy;
		

		GradAll[i].x = (temp < 0) ? 0 : BFP[idx + idy*row + inum*row*column].x;
		GradAll[i].y = (temp < 0) ? 0 : BFP[idx + idy*row + inum*row*column].y;
	

	}
}
///////////////////////
//////////////// Executable functions 
///////////////////////


void ExtractGradients(float* h_rawImg, int* arraySize, int* imgProperties,
	float* h_ImgDxOutRe, float* h_ImgDxOutIm,
	float* h_ImgDyOutRe, float* h_ImgDyOutIm,
	float* h_ImgDCOutRe, float* h_ImgDCOutIm) {

	//Declare constants
	const int row = arraySize[0];
	const int column = arraySize[1];
	const int zrange = 1; // in this case Matz is only doing one image at a time
	const int imgpropsize = arraySize[2];
	const size_t size2Darray = row*column;
	const size_t mem2Darray = size2Darray * sizeof(float);
	const size_t mem2DFFTsize = size2Darray * sizeof(hipfftComplex);

	// Declare all constant regarding the Kernel execution sizes, will need to add a possibility to modify these from the LV as arguments
	const int BlockSizeAll = arraySize[3]; //my computer should be 512
	const int GridSizeKernel = (size2Darray + BlockSizeAll - 1) / BlockSizeAll;

// Copy Raw Img and spatial filtering constants to GPU device
	float* d_rawImg;
	int* d_imgProperties;
	const size_t sizePrp = imgpropsize * sizeof(int);
	hipMalloc((void**)&d_rawImg, mem2Darray);
	hipMemcpy(d_rawImg, h_rawImg, mem2Darray, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_imgProperties, sizePrp);
	hipMemcpy(d_imgProperties, imgProperties, sizePrp, hipMemcpyHostToDevice);
	

//Img memory allocations on the GPU to hold BFP and derivates of X and Y
	hipfftComplex *d_BFP;
	hipfftComplex *d_GradDx;
	hipfftComplex *d_GradDy;
	hipfftComplex *d_DC;
	hipMalloc((void**)&d_BFP, mem2DFFTsize);
	hipMalloc((void**)&d_GradDx, mem2DFFTsize);
	hipMalloc((void**)&d_GradDy, mem2DFFTsize);
	hipMalloc((void**)&d_DC, mem2DFFTsize);
	
	

 /////////////////////////////////////////////////////////////////////////////////////////
 ///// Prepare batch 2D FFT plan, const declaration
 /////////////////////////////////////////////////////////////////////////////////////////
	
	int istride = 1; //means every element is used in the computation
	int ostride = 1; //means every element used in the computatio is output
	int idist = row*column;
	int odist = row*column;
	int inembed[] = { row,column };
	int onembed[] = { row,column };
	const int NRANK = 2;
	int n[NRANK] = { row,column };
	int BATCH = zrange;

	hipfftHandle SingleFFTPlan;
		if (hipfftPlanMany(&SingleFFTPlan, NRANK, n,
		inembed, istride, idist,// *inembed, istride, idist 
		onembed, ostride, odist,// *onembed, ostride, odist 
		HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return;
	}

	

	// Convert d-raw img into a complex number!
		real2complex <<<GridSizeKernel, BlockSizeAll, 0, 0 >>>(d_rawImg, d_BFP, size2Darray);
		hipFree(d_rawImg);

	/// Execute FFT transform in-place to go into kspace, 
		hipfftExecC2C(SingleFFTPlan, d_BFP, d_BFP, HIPFFT_FORWARD);
		//so far so good up to here

	/// Extract gradients in X and Y, frequency filtering 
		
		ExtractGradsBFP << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_BFP, d_GradDx, d_GradDy, d_DC, d_imgProperties, row, column);
		hipFree(d_BFP);
			

	/// Inverse FFT in-place for each of the gradients 
		// can think of batching this!
		hipfftExecC2C(SingleFFTPlan, d_GradDx, d_GradDx, HIPFFT_BACKWARD);
		hipfftExecC2C(SingleFFTPlan, d_GradDy, d_GradDy, HIPFFT_BACKWARD);
		hipfftExecC2C(SingleFFTPlan, d_DC, d_DC, HIPFFT_BACKWARD);

	//free handle , Although might be able to reuse upon the last execution
		hipfftDestroy(SingleFFTPlan);
	
// Copy FFT result to output

		float *d_ImgDxOutRe; 
		float *d_ImgDxOutIm;
		float *d_ImgDyOutRe;
		float *d_ImgDyOutIm;
		float *d_ImgDCOutRe;
		float *d_ImgDCOutIm;
		hipMalloc((void**)&d_ImgDxOutRe, mem2Darray);
		hipMalloc((void**)&d_ImgDxOutIm, mem2Darray);
		hipMalloc((void**)&d_ImgDyOutRe, mem2Darray);
		hipMalloc((void**)&d_ImgDyOutIm, mem2Darray);
		hipMalloc((void**)&d_ImgDCOutRe, mem2Darray);
		hipMalloc((void**)&d_ImgDCOutIm, mem2Darray);
		

		C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_GradDx, d_ImgDxOutRe, d_ImgDxOutIm, size2Darray);
		C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_GradDy, d_ImgDyOutRe, d_ImgDyOutIm, size2Darray);
		C2R << <GridSizeKernel, BlockSizeAll, 0, 0 >> > (d_DC, d_ImgDCOutRe, d_ImgDCOutIm, size2Darray);
		hipFree(d_GradDx);
		hipFree(d_GradDy);
		hipFree(d_DC);
		


		hipMemcpy(h_ImgDxOutRe, d_ImgDxOutRe, mem2Darray, hipMemcpyDeviceToHost);
		hipMemcpy(h_ImgDxOutIm, d_ImgDxOutIm, mem2Darray, hipMemcpyDeviceToHost);
		hipFree(d_ImgDxOutRe);
		hipFree(d_ImgDxOutIm);

		hipMemcpy(h_ImgDyOutRe, d_ImgDyOutRe, mem2Darray, hipMemcpyDeviceToHost);
		hipMemcpy(h_ImgDyOutIm, d_ImgDyOutIm, mem2Darray, hipMemcpyDeviceToHost);
		hipFree(d_ImgDyOutRe);
		hipFree(d_ImgDyOutIm);

		hipMemcpy(h_ImgDCOutRe, d_ImgDCOutRe, mem2Darray, hipMemcpyDeviceToHost);
		hipMemcpy(h_ImgDCOutIm, d_ImgDCOutIm, mem2Darray, hipMemcpyDeviceToHost);
		hipFree(d_ImgDCOutRe);
		hipFree(d_ImgDCOutIm);


		//exporting is correct
		//d_ImgdxOutRe
	///////////
	// FFT ends
	///////////

}


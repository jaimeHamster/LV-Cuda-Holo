#include "hip/hip_runtime.h"
//want to include the feature point detection using templates


//Calculate the sum image, cumsum over all pixels
#include "CudaDLL.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/device_functions.h>
#include <math.h>
#include <float.h>
#include <assert.h>

//Just some definitions, will probably move it to the header files
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;


// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[threadIdx.y + j][threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[(y + j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


__global__ void transposeNoBankConflicts2(float *odata, float *idata, int row, int column)
{
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];

	int xIdx = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIdx = blockIdx.y * TILE_DIM + threadIdx.y;
	
	if ((xIdx < row) && (yIdx < column)) {
		tile[threadIdx.y][threadIdx.x] = idata[(yIdx *row + xIdx];
	}

	__syncthreads();

	xIdx = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	yIdx = blockIdx.x * TILE_DIM + threadIdx.y;

	if ((xIdx < row) && (yIdx < column)) {
		odata[yIdx *column + xIdx] = tile[threadIdx.x][threadIdx.y];
	}
}

__global__ void KernTranspose(float* img_out, float* img_in, int row, int column)
{
	__shared__ float temp[TILE_DIM][TILE_DIM + 1];
	int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;

	if ((xIndex < row) && (yIndex < column)) {
		temp[threadIdx.y][threadIdx.x] = in(xIndex, yIndex);
	}

	__syncthreads();

	xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y;

	if ((xIndex < row ) && (yIndex < column)) {
		out(xIndex, yIndex) = temp[threadIdx.x][threadIdx.y];
	}
}

__global__ void SumImage(float* img2Darray, float* sumimg, int row, int column)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int imgsize = row * column;

	//additional counters
	for (int i = threadID; i < imgsize; i += numThreads)
	{
		int idx = k % row;
		int idy = k / row;
		




	}
}



const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

int main(int argc, char **argv)
{
	const int nx = 1024;
	const int ny = 1024;
	const int mem_size = nx * ny * sizeof(float);

	dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

	int devId = 0;
	if (argc > 1) devId = atoi(argv[1]);

	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, devId));
	printf("\nDevice : %s\n", prop.name);
	printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n",
		nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
	printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
		dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

	checkCuda(hipSetDevice(devId));

	float *h_idata = (float*)malloc(mem_size);
	float *h_cdata = (float*)malloc(mem_size);
	float *h_tdata = (float*)malloc(mem_size);
	float *gold = (float*)malloc(mem_size);

	float *d_idata, *d_cdata, *d_tdata;
	checkCuda(hipMalloc(&d_idata, mem_size));
	checkCuda(hipMalloc(&d_cdata, mem_size));
	checkCuda(hipMalloc(&d_tdata, mem_size));

	// check parameters and calculate execution configuration
	if (nx % TILE_DIM || ny % TILE_DIM) {
		printf("nx and ny must be a multiple of TILE_DIM\n");
		goto error_exit;
	}

	if (TILE_DIM % BLOCK_ROWS) {
		printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
		goto error_exit;
	}

	// host
	for (int j = 0; j < ny; j++)
		for (int i = 0; i < nx; i++)
			h_idata[j*nx + i] = j * nx + i;

	// correct result for error checking
	for (int j = 0; j < ny; j++)
		for (int i = 0; i < nx; i++)
			gold[j*nx + i] = h_idata[i*nx + j];

	// device
	checkCuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

	// events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// ------------
	// time kernels
	// ------------
	printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");

	// ----
	// copy 
	// ----
	printf("%25s", "copy");
	checkCuda(hipMemset(d_cdata, 0, mem_size));
	// warm up
	copy << <dimGrid, dimBlock >> > (d_cdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		copy << <dimGrid, dimBlock >> > (d_cdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(h_idata, h_cdata, nx*ny, ms);

	// -------------
	// copySharedMem 
	// -------------
	printf("%25s", "shared memory copy");
	checkCuda(hipMemset(d_cdata, 0, mem_size));
	// warm up
	copySharedMem << <dimGrid, dimBlock >> > (d_cdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		copySharedMem << <dimGrid, dimBlock >> > (d_cdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(h_idata, h_cdata, nx * ny, ms);

	// --------------
	// transposeNaive 
	// --------------
	printf("%25s", "naive transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	// warmup
	transposeNaive << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeNaive << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);

	// ------------------
	// transposeCoalesced 
	// ------------------
	printf("%25s", "coalesced transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	// warmup
	transposeCoalesced << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeCoalesced << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);

	// ------------------------
	// transposeNoBankConflicts
	// ------------------------
	printf("%25s", "conflict-free transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	// warmup
	transposeNoBankConflicts << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeNoBankConflicts << <dimGrid, dimBlock >> > (d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);

error_exit:
	// cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(d_tdata));
	checkCuda(hipFree(d_cdata));
	checkCuda(hipFree(d_idata));
	free(h_idata);
	free(h_tdata);
	free(h_cdata);
	free(gold);
}

inline __device__
void PrefixSum(Tout* output, Tin* input, int w, int nextpow2)
{
	SharedMemory<Tout> shared;
	Tout* temp = shared.getPointer();

	const int tdx = threadIdx.x;
	int offset = 1;
	const int tdx2 = 2 * tdx;
	const int tdx2p = tdx2 + 1;

	temp[tdx2] = tdx2 < w ? input[tdx2] : 0;
	temp[tdx2p] = tdx2p < w ? input[tdx2p] : 0;

	for (int d = nextpow2 >> 1; d > 0; d >>= 1) {
		__syncthreads();
		if (tdx < d)
		{
			int ai = offset * (tdx2p)-1;
			int bi = offset * (tdx2 + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (tdx == 0) temp[nextpow2 - 1] = 0;

	for (int d = 1; d < nextpow2; d *= 2) {
		offset >>= 1;

		__syncthreads();

		if (tdx < d)
		{
			int ai = offset * (tdx2p)-1;
			int bi = offset * (tdx2 + 2) - 1;
			Tout t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	if (tdx2 < w)  output[tdx2] = temp[tdx2];
	if (tdx2p < w) output[tdx2p] = temp[tdx2p];
}

template<typename Tout, typename Tin>
__global__ void KernPrefixSumRows(Image<Tout> out, Image<Tin> in)
{
	const int row = blockIdx.y;
	PrefixSum<Tout, Tin>(out.RowPtr(row), in.RowPtr(row), in.w, 2 * blockDim.x);
}

template<typename Tout, typename Tin>
void PrefixSumRows(Image<Tout> out, Image<Tin> in)
{
	dim3 blockDim = dim3(1, 1);
	while (blockDim.x < ceil(in.w / 2.0f)) blockDim.x <<= 1;
	const dim3 gridDim = dim3(1, in.h);
	KernPrefixSumRows << <gridDim, blockDim, 2 * sizeof(Tout)*blockDim.x >> > (out, in);
}
